#include "hip/hip_runtime.h"
#include <stdio.h>

//KERNEL CODE
//Executed by multiple threads in parallell
__global__
void saxpy(int n, float a, float *x, float *y){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) y[i] = a*x[i] + y[i];
}


int main(void){
	int N = 1<<20;
	float *x, *y, *d_x, *d_y;
	//the variables x and y points to the host arrays
	x = (float*)malloc(N*sizeof(float));
	y = (float*)malloc(N*sizeof(float));

	hipMalloc(&d_x, N*sizeof(float)); 
	hipMalloc(&d_y, N*sizeof(float));

	//The host code then initializes the host arrays
	for(int i=0; i<N; i++){
		x[i] = 1.0f	//Sets x to an array of 1s
		y[i] = 2.0f	//Sets y to an array of 2s
	}

	//Copy the data from x and y to the corresponding device arrays d_x and d_y
	hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

        // Launch the kernel
        // Perform SAXPY on 1M elements
        saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

	//Copy result back to host
	hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
    		maxError = max(maxError, abs(y[i]-4.0f));
  	printf("Max error: %f\n", maxError);

	//Cleaning up
	hipFree(d_x);
  	hipFree(d_y);
  	free(x);
  	free(y);
}
