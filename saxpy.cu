#include <stdio.h>

float *x, *y, *d_x, *d_y;
//the variables x and y points to the host arrays
x = (float*)malloc(N*sizeof(float));
y = (float*)malloc(N*sizeof(float));

hipMalloc(&d_x, N*sizeof(float)); 
hipMalloc(&d_y, N*sizeof(float));

//The host code then initializes the host arrays
for(int i=0; i<N; i++){
x[i] = 1.0f	//Sets x to an array of 1s
y[i] = 2.0f	//Sets y to an array of 2s
}

//Copy the data from x and y to the corresponding device arrays d_x and d_y
hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

//Copy result back to host
hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
